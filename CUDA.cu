#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <cstring>

using namespace std;

 struct matriz {
    
       int N1, M1;
       int total;
       double* ptr;
    
       matriz(int _N1, int _M1): N1(_N1), M1(_M1), total(_N1*_M1) {
        ptr = new double[_N1*_M1];
        }
    
        ~matriz() {
        delete[] ptr;
        }
    
        double* operator[](int i) {
        return &(ptr[i*M1]);
        }
        };

void hora(char data[80])
  { 
  time_t rawtime;
  struct tm * timeinfo;
  char data_now [80];
  time (&rawtime);
  timeinfo = localtime (&rawtime);
  strftime (data_now,80,"%d-%m--%H-%M-%S",timeinfo);
  puts (data_now);
  sprintf(data,"%s",data_now);
  }

#define c 299792458
#define pi 3.1415926535897932384626433832795
  
#define tn 1e-12                            //Tempo de normalizacao do programa - unidade de t' (unidade de tempo do prog.) = tn segundos
#define dt 0.2 								//Passo temporal em unidades de t' 

#define Go1 3.2e3
#define No1 1.5e8
#define eps1 5.0e-7
#define gammap1 282.0e9  // gammap = (wox-woy)/2  //3.20347e9
#define gammas1 0.5e9
#define alpha1 3.0
#define kappa1 11e9
#define kappa2 11e9

#define wo 0.0 //2.5132741228718345907701147066236e15 //2.217e15

__device__ fun1 (double x1, double y1, double z1, double RE1delay,double RE1delay2) // cos1,cos2,sin1,sin2
{
       return (0.5*tn*((Go1*(z1-No1)/(1.0 + eps1*(x1*x1 + y1*y1))) - gammap1)*(x1-(alpha1*y1)) + 
       tn*kappa1*RE1delay+ tn*kappa2*RE1delay2);
}

__device__ fun2 (double x1, double y1, double z1, double IE1delay, double IE1delay2) // RE1delay,RE1delay2
{
       return (0.5*tn*((Go1*(z1-No1)/(1.0 + eps1*(x1*x1 + y1*y1))) - gammap1)*((alpha1*x1)+y1) + 
       tn*kappa1*IE1delay + tn*kappa2*IE1delay2);
}

__device__ fun3 (double x1, double y1, double z1,double J1)
{
       return (tn*(J1-(z1*gammas1)-((Go1*(z1-No1)/(1.0 + eps1*(x1*x1 + y1*y1))))*(x1*x1 + y1*y1)));
}

__global__ RK (,)
{
	double k11,k12,k13,k21,k22,k23,k31,k32,k33,k41,k42,k43;
	//ATENÇÃO: LAÇO "FOR" DO TRANSIENTE TRANSIENTE//
for (l=0;l<itransiente+1;l=l+1)
{
l1=l1+1;  
		
//MÉTODO RUNGE KUTTA 4ª ORDEM


        k11=dt*fun1(x1[0],x1[1],x1[2],RE1delay,RE1delay2); //
		k12=dt*fun2(x1[0],x1[1],x1[2],IE1delay,IE1delay2); //
		k13=dt*fun3(x1[0],x1[1],x1[2],J1); //

		k21=dt*fun1(x1[0]+k11/2.0,x1[1]+k12/2.0,x1[2]+k13/2.0,RE1delay,RE1delay2); //
		k22=dt*fun2(x1[0]+k11/2.0,x1[1]+k12/2.0,x1[2]+k13/2.0,IE1delay,IE1delay2); //
		k23=dt*fun3(x1[0]+k11/2.0,x1[1]+k12/2.0,x1[2]+k13/2.0,J1);

		k31=dt*fun1(x1[0]+k21/2.0,x1[1]+k22/2.0,x1[2]+k23/2.0,RE1delay,RE1delay2); //
		k32=dt*fun2(x1[0]+k21/2.0,x1[1]+k22/2.0,x1[2]+k23/2.0,IE1delay,IE1delay2); //
		k33=dt*fun3(x1[0]+k21/2.0,x1[1]+k22/2.0,x1[2]+k23/2.0,J1);

		k41=dt*fun1(x1[0]+k31, x1[1]+k32, x1[2]+k33,RE1delay,RE1delay2); //
		k42=dt*fun2(x1[0]+k31, x1[1]+k32, x1[2]+k33,IE1delay,IE1delay2); //
		k43=dt*fun3(x1[0]+k31, x1[1]+k32, x1[2]+k33,J1);

		/*Eaa=Ea; 	    Ea=x[0]; 	    Iaa=Ia; 	    Ia=I;*/
		x1[0]+=(k11+2*k21+2*k31+k41)/6.0;
		x1[1]+=(k12+2*k22+2*k32+k42)/6.0;
		x1[2]+=(k13+2*k23+2*k33+k43)/6.0;
   
	RE1[h11][0]=x1[0];
	IE1[h11][0]=x1[1];
	RE2[h12][0]=x1[0];
	IE2[h12][0]=x1[1];	
  
    RE1delay=RE1[h21][0];
    IE1delay=IE1[h21][0];
    RE1delay2=RE2[h22][0];
    IE1delay2=IE2[h22][0];
    
    h11=h11+1;
    h21=h21+1;
    h12=h12+1;
    h22=h22+1;

        if(h11==idelay1+1)
        {
                      h11=0;
        }
        
         if(h21==idelay1+1)
        {
                      h21=0;
        }
	
	 	if(h12==idelay2+1)
        {
                      h12=0;
        }
        
         if(h22==idelay2+1)
        {
                      h22=0;
        }	

    I1= x1[0]*x1[0]+x1[1]*x1[1];        
    t=t+dt;
 
    Ij=(I1+Ij_1*tfilter*exp(-1.0/tfilter))/tfilter;                                                          
    Ij_1=Ij;
    }

for (l=itransiente;l<itotal+1;l=l+1)
{
    
l1=l1+1;  
		
//MÉTODO RUNGE KUTTA 4ª ORDEM

     //******LASER 1******// k1[0]=k11,k1[1]=k12,k1[2]=k13, k-Tipo do k-Equação a que pertence
     
        //+eta*(x1[2]-x2[2]);

        k11=dt*fun1(x1[0],x1[1],x1[2],RE1delay,RE1delay2); //
		k12=dt*fun2(x1[0],x1[1],x1[2],IE1delay,IE1delay2); //
		k13=dt*fun3(x1[0],x1[1],x1[2],J1); //

		k21=dt*fun1(x1[0]+k11/2.0,x1[1]+k12/2.0,x1[2]+k13/2.0,RE1delay,RE1delay2); //
		k22=dt*fun2(x1[0]+k11/2.0,x1[1]+k12/2.0,x1[2]+k13/2.0,IE1delay,IE1delay2); //
		k23=dt*fun3(x1[0]+k11/2.0,x1[1]+k12/2.0,x1[2]+k13/2.0,J1);

		k31=dt*fun1(x1[0]+k21/2.0,x1[1]+k22/2.0,x1[2]+k23/2.0,RE1delay,RE1delay2); //
		k32=dt*fun2(x1[0]+k21/2.0,x1[1]+k22/2.0,x1[2]+k23/2.0,IE1delay,IE1delay2); //
		k33=dt*fun3(x1[0]+k21/2.0,x1[1]+k22/2.0,x1[2]+k23/2.0,J1);

		k41=dt*fun1(x1[0]+k31, x1[1]+k32, x1[2]+k33,RE1delay,RE1delay2); //
		k42=dt*fun2(x1[0]+k31, x1[1]+k32, x1[2]+k33,IE1delay,IE1delay2); //
		k43=dt*fun3(x1[0]+k31, x1[1]+k32, x1[2]+k33,J1);

		/*Eaa=Ea; 	    Ea=x[0]; 	    Iaa=Ia; 	    Ia=I;*/
		x1[0]+=(k11+2*k21+2*k31+k41)/6.0;
		x1[1]+=(k12+2*k22+2*k32+k42)/6.0;
		x1[2]+=(k13+2*k23+2*k33+k43)/6.0;
		
    
    
    
	RE1[h11][0]=x1[0];
	IE1[h11][0]=x1[1];
	RE2[h12][0]=x1[0];
	IE2[h12][0]=x1[1];	
  
    RE1delay=RE1[h21][0];
    IE1delay=IE1[h21][0];
    RE1delay2=RE2[h22][0];
    IE1delay2=IE2[h22][0];
    
    h11=h11+1;
    h21=h21+1;
    h12=h12+1;
    h22=h22+1;
    
	
        if(h11==idelay1+1)
        {
                      h11=0;
        }
        
         if(h21==idelay1+1)
        {
                      h21=0;
        }
	
	 	if(h12==idelay2+1)
        {
                      h12=0;
        }
        
         if(h22==idelay2+1)
        {
                      h22=0;
        }	
		


    I1= x1[0]*x1[0]+x1[1]*x1[1];
 //   N1 = x1[2];
 //   J11= J1;
	
	
	
    t=t+dt;
    
    Ij=(I1+Ij_1*tfilter*exp(-1.0/tfilter))/tfilter;
    
    Isum=Isum+I1;
    	                                                                            
     if(l1>2000000000&&((l1%jpp)==0))                        
     {                                                                                                                                             
     l1=jpp;                                                                                                                                       
     }                                                       
    
    //if (((long)(l1%jpp)==0)&&(l1>(int)((1000*1E-9/tn)/dt))) // && l1>1000ns(transiente) 
    //if (((long)(l1%jpp)==0)) // && l1>1000ns(transiente) 
    if (((long)(l1%jpp)==0)) 
			{
			    fprintf(p1,"%lf\t%lf\n",t*tn/1e-9,Ij); // Tempo plotado em nanosegundos

			}
    Ij_1=Ij;
    //fprintf(p1,"%lf\t%lf\t%lf\t%lf\t%lf\t%lf\n",x1[0]*x1[0]+x1[1]*x1[1],x2[0]*x2[0]+x2[1]*x2[1],x1[2],x2[2],t*tn/1e-9,((x1[0]*x1[0]+x1[1]*x1[1])-(x2[0]*x2[0]+x2[1]*x2[1])));

}
}

int main(int argc, char *argv[])
{

	char data [80];
	hora(data);
	
	// DEFINICOES DE COMANDOS NO WINDOWS
	char fazerdiretorio[] = "mkdir";
    char mover[] = "move";
    char copiar[]= "copy";
	
	//PONTEIROS PARA PEGAR NOME DO PROGRAMA
	char *ptr1, *ptr2, *ptr3, *ptr4;
	ptr2 = strtok (argv[0], "\\"); // pointer to first "token" in argv[0]
    while(ptr2 != NULL) { // while current is not null
        ptr1 = ptr2; // copy current to previous
        ptr2 = strtok(NULL, "\\"); // find next
    } 
	   
    ptr2=ptr1;
    ptr2 = strstr (ptr1, ".exe"); // pointer to first "token" in argv[0]
	strncpy(ptr2,".cpp",4); 
	
	//IMPRIMIR NOME DO PROGRAMA
	char formatNomePrograma[]="%s";
	char nome_programa[300];
	sprintf(nome_programa,formatNomePrograma,ptr1);	
	cout<< nome_programa << endl;

	char formatnome_programa2[]="%s-%s";
	char nome_programa2[300];
	sprintf(nome_programa2,formatnome_programa2,data,nome_programa);
		
	char fileNomePrograma[300];
	sprintf(fileNomePrograma,"%s %s %s",copiar,nome_programa,nome_programa2);
	system(fileNomePrograma);
    
    // CRIACAO DA PASTA 0
    strncpy(ptr2,"\0",1);
    char formatNomePasta[]="%s";
	char nome_pasta[200];
	sprintf(nome_pasta,formatNomePasta,ptr1);	
    
    char pasta0[200];   
    sprintf(pasta0,"%s_%s",data,nome_pasta);
    char file00[200];
    sprintf(file00,"%s %s",fazerdiretorio,pasta0);
    system(file00);
    
    //COPIAR UMA COPIA DO PROGRAMA PARA A PASTA 0 //

	char fileNomePrograma2[300];
	sprintf(fileNomePrograma2,"%s %s %s",mover,nome_programa2,pasta0);
	system(fileNomePrograma2);


// DECLARAÇÕES DAS VARIÁVEIS
double t,ttotalreal,ttotal,tmem,transiente,J1,Jo,RE1ret,IE1ret,RE2ret,IE2ret;
int i,j,k,m1,m2,h11,h21,h12,h22,lmax,l1,jpp,cont,tfilter;
int idelay1,idelay2,itransiente;
double itotal,l,Lfeed1,Lfeed2,Lrelativo,Tmed,Tvar,Tdesv,TmedProv,NLFF,ps_p_pontos;
double taudelay1,taudelay2;
//int itotal2;
double taudelay1in, taudelay1fim, deltataudelay1;
// DEFINIÇÕES DOS PARÂMETROS


ttotalreal = 2000000;                     //Tempo de iteração em nanosegundos
ps_p_pontos= 20000;                        //pegar pontos a cada ps_p_pontos pico segundos
tmem = 10000;                               // em unidades de tn (de acordo com a unidade temporal da série bruta calculada. Geralmente em ps)
transiente= 10000; 
tfilter = (int)(tmem/dt);
lmax= (int) tfilter+1;
itransiente=(int)(((transiente*1E-9/tn)/dt)+1);                               //  em unidades de tn   //  ESCOLHER 1ns      Fonte: Programa de John
jpp = (int) ((ps_p_pontos/dt)+1);                                        // para dt=0.5 ps, jpp=20 (pegar ponto a cada 10ps)

taudelay2 =50.0e-9; 									// Tempo de ida e volta na cavidade - Em unidades de segundos // 

taudelay1in = 0.4946*taudelay2;
taudelay1fim= 0.49461*taudelay2;
deltataudelay1=0.00002*taudelay2;

taudelay1 = taudelay1in;

//condições iniciais

double RE1i,RE2i,IE1i,IE2i,N1i,N2i;

RE1i= 1;
IE1i= 1.01;
N1i=  1e8;

//ATENÇÃO: Os valores abaixo NÃO SÃO ARBITRÁRIOS e devem ser calculados com os parâmetros acima

ttotal = (ttotalreal*1E-9)/tn;                  //Tempo de iteração em unidades de t'   
itotal = ceil((ttotal/dt)+1);             	        //Nº total de iterações  
//itotal2 = (int) (itotal);

idelay2 = (int) (((taudelay2/tn)/dt)+1);            //Nº de iterações para feedback laser1  

int l2=0;

FILE *p2;
char filename2[200]; 
char format2[]="%s-LS-filtrada-t1_%.4lf-Lrelativo.txt";
sprintf(filename2,format2,data,taudelay1*1e9);
p2=fopen(filename2,"w"); 

char fileMovp2[200];
sprintf(fileMovp2,"%s %s %s",mover,filename2,pasta0); 


FILE *p2_1;
char filename2_1[200]; 
char format2_1[]="%s-Ibar-filtrada-t1_%.4lf-Lrelativo.txt";
sprintf(filename2_1,format2_1,data,taudelay1*1e9);
p2_1=fopen(filename2_1,"w"); 


char fileMovp2_1[200];
sprintf(fileMovp2_1,"%s %s %s",mover,filename2_1,pasta0); 

//0.17655 - 0.00535
//for(taudelay1=29.527e-9;taudelay1<29.5275e-9;taudelay1=taudelay1+0.001e-9)
//for(taudelay1=26.3541e-9;taudelay1<26.354101e-9;taudelay1=taudelay1+0.00535e-9)
//for(taudelay1=5e-9;taudelay1<=taudelay2;taudelay1=taudelay1+1e-9)
//for(taudelay1=taudelay2*(0.5+pi/300);taudelay1<=taudelay2+1e-9;taudelay1=taudelay1+1e-9)
//{

double Ibar=0, Isum=0;
	
for(taudelay1=taudelay1in;taudelay1<taudelay1fim;taudelay1=taudelay1+deltataudelay1)
{
                                                             
l2=l2+1;

idelay1 = ceil(((taudelay1/tn)/dt)+1);            //Nº de iterações para feedback laser1

Lfeed1=c*taudelay1/2;
Lfeed2=c*taudelay2/2;

Lrelativo=Lfeed1/Lfeed2;  

int dif_idelay;
dif_idelay = idelay1-idelay2;

//cout << "idelay1= " << idelay1 <<endl;
//cout << "idelay2= " << idelay2 <<endl;
//cout << "Lrelativo= " <<Lrelativo  <<endl;



matriz RE1(idelay1+1,1); 
matriz IE1(idelay1+1,1); 
matriz RE2(idelay2+1,1); 
matriz IE2(idelay2+1,1);

double RE1delay;
double IE1delay;
double RE1delay2;
double IE1delay2;


double I1;

//double N1[lmax+1];
//double N2[lmax+1];
//double J11[lmax+1];
//double J22[lmax+1];

// DEFINIÇÕES DOS PARÂMETROS 1


double Nth=No1+(gammap1/(Go1));
double Jth=Nth*gammas1;

//Jo=8e8*1e9;
J1=1.02*Jth;
//cout << "Jo= " << Jo <<endl;
//cout << "Nth= " << Nth <<endl;
//cout << "Jth= " << Jth <<endl;
						
// INICIANDO O PROGRAMA

//data quando o programa começou a rodar ==> para incluir no nome do arquivo
time_t rawtime;
  struct tm * timeinfo;
  char data [80];

  time (&rawtime);
  timeinfo = localtime (&rawtime);

  strftime (data,80,"%d-%m--%H-%M-%S",timeinfo);
  puts (data);
 
  
FILE *p1,*p3;

char format1[]="%s-LS-filtrada-t1_%.4lf.txt";
char filename1[200];
sprintf(filename1,format1,data,taudelay1*1e9);
p1=fopen(filename1,"w"); 

//char s1[] = "Filtro_de_serie-Laser_solitario-parte2";
//char s2[100];
//sprintf(s2,filename1);

char c1[] = "Tlff-hist-dtlff-NumLffBursts-19-03-2018";
//char c2[200];
//sprintf(c2,"filtroLS-%s",filename1);

char d2[300],d3[300],d4[300],d5[300];
sprintf(d2,"TLFF-%s",filename1);
sprintf(d3,"Est-%s",filename1);
sprintf(d4,"dTLFF-%s",filename1);
sprintf(d5,"#LFFs-Burst-%s",filename1);



char fazerdiretorio[] = "mkdir";
char pasta[100];
sprintf(pasta,"tau1-%lf-%s",taudelay1*1e9,data); 

char mover[] = "move";
char del[] = "del";
char destino[100];
sprintf(destino,"%s",pasta);

char file0[100];
sprintf(file0,"%s %s",fazerdiretorio,pasta);

//char file1[500];
//sprintf(file1,"%s %s",s1,s2);

char file2[500];
sprintf(file2,"%s %s",c1,filename1);

char file6[200];
sprintf(file6,"%s %s %s",mover,filename1,destino);

char filedelp1[200];
sprintf(filedelp1,"%s %s /q",del,filename1);

char file7[200];
sprintf(file7,"%s %s %s",mover,d2,destino);

char file8[200];
sprintf(file8,"%s %s %s",mover,d3,destino);

char file9[200];
sprintf(file9,"%s %s %s",mover,d4,destino);

char file9_2[200];
sprintf(file9_2,"%s %s %s",mover,d5,destino);

char fileMovDestino[200];
sprintf(fileMovDestino,"%s %s %s",mover,destino,pasta0);

//cout << "oi5";

//double k11[3], k12[3], k13[3], k14[3];
double x1[3];


//condições iniciais

x1[0]=RE1i; x1[1]=IE1i; x1[2]=N1i;

for(m1=0;m1<idelay1+1;m1++)
{
	RE1[m1][0]=RE1i;
	IE1[m1][0]=IE1i;
}
m1=0;

for(m1=0;m1<idelay2+1;m1++)
{
   	RE2[m1][0]=RE1i;
	IE2[m1][0]=IE1i;
}

t=0.0;

RE1delay=RE1i;
IE1delay=IE1i;
RE1delay2=RE1i;
IE1delay2=IE1i;

    I1= x1[0]*x1[0]+x1[1]*x1[1];
 //   N1[lmax]= x1[2];
 //   J11[lmax]=0;
 //   N2[lmax]= x2[2];
 //   J22[lmax]=0;


h11=idelay1-1;
h21=0;

h12=idelay2-1;
h22=0;

l1=0;

double Ij,Ij_1;
Ij=I1;
Ij_1=I1;


Ibar=Isum/(itotal-itransiente);
cout<<"L1/L2: " <<Lrelativo <<"\t" <<"Ibar: " <<Ibar << endl;
fprintf(p2_1,"%lf\t%lf\n",Lrelativo,Ibar);
Ibar=0;
Isum=0;
//cout << "Calculo feito\n";

fclose(p1);

system(file0); // criar pasta

//cout << endl <<endl <<"ABRINDO PROGRAMA: " <<s1 <<endl;
//system(file1);

cout << endl <<endl <<"ABRINDO PROGRAMA: " <<c1 <<endl;
system(file2);
cout << endl;

//system(file4);
system(file6);

//system(filedelp1);

system(file7);

FILE *leitorEST;
leitorEST=fopen(d3,"r"); 

    for (cont=0;cont<1;cont++)
	{
	char ignore1[1024];
    fgets(ignore1, sizeof(ignore1), leitorEST);
    }
              
   	for (cont=0;cont<1;cont++)
	{
        fscanf(leitorEST,"%lf",&NLFF);      //  coluna 1 do arquivo 1       
		fscanf(leitorEST,"%lf",&Tmed);     //  coluna 2 do arquivo 1 
		fscanf(leitorEST,"%lf",&TmedProv);     //  coluna 2 do arquivo 1 
        fscanf(leitorEST,"%lf",&Tvar);      //  coluna 1 do arquivo 1       
		fscanf(leitorEST,"%lf",&Tdesv);
          
    }

fclose(leitorEST);

system(file8);
system(file9);
system(file9_2);
system(fileMovDestino);

fprintf(p2,"%lf\t%lf\t%lf\t%lf\t%lf\t%lf\n",Lrelativo,Tmed,TmedProv,Tvar,Tdesv,NLFF);

}

fclose(p2);
system(fileMovp2);

fclose(p2_1);
system(fileMovp2_1);


hora(data);
//fclose(p2);
system("PAUSE");
return 0 ;


}
     
